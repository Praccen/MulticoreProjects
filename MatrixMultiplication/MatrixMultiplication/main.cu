#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__
void matrixMult(float *N, float *M, float *P, int Size) { //Client
	int threadIndex = threadIdx.x;
	int blockIndex = blockIdx.x;
	int blockSize = blockDim.x;
	int stride = gridDim.x * blockSize;

	for (int i = threadIndex + blockIndex * blockSize; i < Size * Size; i += stride) {

	}
}

void main() { //Host
//float N[Size][Size], M[Size][Size], P[Size][Size]; //hipMalloc?
	float *N_d, *M_d, *P_d;
	int allocSize = Size * Size * sizeof(float);

	GetMatrix(N, file1); GetMatrix(M, file2); /* Read N and M */

	hipMalloc((void **), &N_d, allocSize);
	hipMemcpy(N_d, N, allocSize, hipMemcpyHostToDevice);

	hipMalloc((void **), &M_d, allocSize);
	hipMemcpy(M_d, M, allocSize, hipMemcpyHostToDevice);

	hipMalloc((void **), &P_d, allocSize);

	int numberOfBlocks = 4;
	int numberOfThreads = 32;
	matrixMult << <numberOfBlocks, numberOfThreads >> > (N_d, M_d, P_d, Size);

	hipMemcpy(P, P_d, allocSize, hipMemcpyDeviceToHost);

	PutMatrix(P, file3);  /* Skriv ut P */

	hipFree(N_d);
	hipFree(M_d);
	hipFree(P_d);
}
