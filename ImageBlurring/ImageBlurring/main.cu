#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "CImg.h"

using Clock = std::chrono::steady_clock;
using std::chrono::time_point;
using std::chrono::duration_cast;
using std::chrono::milliseconds;

using namespace cimg_library;

void blurSeq() {

	// Don't use the small cake 
	// CImg<unsigned char> image("cake-small.ppm"), blurimage("cake-small.ppm");

	// Use the big cake
	CImg<unsigned char> image("cake.ppm"), blurimage("cake.ppm");

	// Don't use the small mask
	/* Create a mask of weights (3 x 3 Gaussian blur)
	double a3 = 1.0 / 16.0;
	double b3 = 2.0 / 16.0;
	double c3 = 4.0 / 16.0;
	CImg<> mask3 = CImg<>(3, 3).fill(
		a3, b3, a3,
		b3, c3, b3,
		a3, b3, a3);
	 */

	 // Use the big mask
	 // Create the mask of weights (5 x 5 Gaussian blur)
	CImg<double> mask5(5, 5);
	mask5(0, 0) = mask5(0, 4) = mask5(4, 0) = mask5(4, 4) = 1.0 / 256.0;
	mask5(0, 1) = mask5(0, 3) = mask5(1, 0) = mask5(1, 4) = mask5(3, 0) = mask5(3, 4) = mask5(4, 1) = mask5(4, 3) = 4.0 / 256.0;
	mask5(0, 2) = mask5(2, 0) = mask5(2, 4) = mask5(4, 2) = 6.0 / 256.0;
	mask5(1, 1) = mask5(1, 3) = mask5(3, 1) = mask5(3, 3) = 16.0 / 256.0;
	mask5(1, 2) = mask5(2, 1) = mask5(2, 3) = mask5(3, 2) = 24.0 / 256.0;
	mask5(2, 2) = 36.0 / 256.0;

	// Print the mask that is being used. Note: Doesn't look so pretty.
	for (int i = 0; i <= 4; i++)
	{
		for (int j = 0; j <= 4; j++)
			std::cout << mask5(i, j) << " ";
		std::cout << "\n";
	}

	// Convolve and record the time taken to do the operation
	auto begin = std::chrono::high_resolution_clock::now();
	// Blur the image!
	blurimage.convolve(mask5);
	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - begin;
	std::cout << "Time taken to convolve = " << elapsed.count() << " seconds";

	// Show the original and the blurred images and compare.

	// To display the images as 400 x 300
	/*
	CImgDisplay main_disp(400, 300, "Original image");
	CImgDisplay main_disp2(400, 300, "Blurred image");
	main_disp.render(image);
	main_disp2.render(blurimage);
	*/

	// Display the images in their original size
	CImgDisplay main_disp(image, "Original image");
	CImgDisplay main_disp2(blurimage, "Blurred image");

	getchar();

	/*while (1)
	{
		main_disp.wait(); main_disp2.wait();
	}*/
}

__global__
void blurParClient(float *imageOrig, float *imageBlurred, int imageWidth, int imageHeight, float *mask) {
	int threadIndex = threadIdx.x;
	int blockIndex = blockIdx.x;
	int blockSize = blockDim.x;
	int stride = gridDim.x * blockSize;

	//i * imageHeight + j * 3 + k

	for (int i = threadIndex + blockIndex * blockSize; i < imageWidth * imageHeight; i += stride) {
		int x = i % imageWidth;
		int y = (int)floor((double)(i / imageWidth));

		for (int c = 0; c < 3; c++) {
			float tempValue = 0;
			for (int maskY = -2; maskY < 3; maskY++) {
				for (int maskX = -2; maskX < 3; maskX++) {
					if (x + maskX >= 0 && x + maskX < imageWidth && y + maskY >= 0 && y + maskY < imageHeight) {
						tempValue += imageOrig[(y + maskY) * imageWidth * 3 + (x + maskX) * 3 + c] * mask[(maskY + 2) * 5 + (maskX + 2)];
					}
				}
			}
			imageBlurred[y * imageWidth * 3 + x * 3 + c] = tempValue;
		}
	}
}

void blurParHost() {
	//Use CImg library to load the image(s) and mask
	CImg<unsigned char> image("cake-small.ppm"), blurimage("cake-small.ppm");

	CImg<double> mask5(5, 5);
	mask5(0, 0) = mask5(0, 4) = mask5(4, 0) = mask5(4, 4) = 1.0 / 256.0;
	mask5(0, 1) = mask5(0, 3) = mask5(1, 0) = mask5(1, 4) = mask5(3, 0) = mask5(3, 4) = mask5(4, 1) = mask5(4, 3) = 4.0 / 256.0;
	mask5(0, 2) = mask5(2, 0) = mask5(2, 4) = mask5(4, 2) = 6.0 / 256.0;
	mask5(1, 1) = mask5(1, 3) = mask5(3, 1) = mask5(3, 3) = 16.0 / 256.0;
	mask5(1, 2) = mask5(2, 1) = mask5(2, 3) = mask5(3, 2) = 24.0 / 256.0;
	mask5(2, 2) = 36.0 / 256.0;

	int imageWidth = image.width();
	int imageHeight = image.height();

	//Allocate
	float *imageOrig;
	hipMallocManaged(&imageOrig, imageWidth * imageHeight * 3 * sizeof(float));

	float *imageBlurred;
	hipMallocManaged(&imageBlurred, imageWidth * imageHeight * 3 * sizeof(float));

	float *mask;
	hipMallocManaged(&mask, 5 * 5 * sizeof(float));


	//Get image values
	for (int y = 0; y < imageHeight; y++) {
		for (int x = 0; x < imageWidth; x++) {
			for (int c = 0; c < 3; c++) {
				imageOrig[y * imageWidth * 3 + x * 3 + c] = (float)image(x, y, 0, c);
			}
		}
	}

	//Get mask values
	for (int y = 0; y < 5; y++) {
		for (int x = 0; x < 5; x++) {
			mask[y * 5 + x] = mask5(x, y);
		}
	}


	int nrOfBlocks = 32;
	int nrOfThreadsPerBlock = 1024;

	blurParClient << <nrOfBlocks, nrOfThreadsPerBlock >> > (imageOrig, imageBlurred, imageWidth, imageHeight, mask);
	hipDeviceSynchronize();

	//Set image values
	for (int y = 0; y < imageHeight; y++) {
		for (int x = 0; x < imageWidth; x++) {
			for (int c = 0; c < 3; c++) {
				blurimage(x, y, 0, c) = imageBlurred[y * imageWidth * 3 + x * 3 + c];
			}
		}
	}

	// Display the images in their original size
	CImgDisplay main_disp(image, "Original image");
	CImgDisplay main_disp2(blurimage, "Blurred image");

	getchar();

	//Deallocate
	if (hipFree(imageOrig) != hipSuccess) {
		std::cout << "Could not free imageOrig\n";
	}
	if (hipFree(imageBlurred) != hipSuccess) {
		std::cout << "Could not free imageBlurred\n";
	}
	if (hipFree(mask) != hipSuccess) {
		std::cout << "Could not free mask\n";
	}
}

int main() {

	std::cout << "Hello world!\n\n";

	//blurSeq();

	blurParHost();

	hipDeviceReset();
	return 0;
}