#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using Clock = std::chrono::steady_clock;
using std::chrono::time_point;
using std::chrono::duration_cast;
using std::chrono::milliseconds;

//Options
int g_matrixSize;
const char* g_init;
int g_maxNum;

double **initMatrixSeq() {
	double **matrix = new double*[g_matrixSize];

	for (int i = 0; i < g_matrixSize; i++) {
		matrix[i] = new double[g_matrixSize];
	}

	if (strcmp(g_init, "rand") == 0) {
		for (int i = 0; i < g_matrixSize; i++) {
			for (int j = 0; j < g_matrixSize; j++) {
				if (i == j) { //diagonal dominance
					matrix[i][j] = (double)(rand() % g_maxNum) + 5.0;
				}
				else {
					matrix[i][j] = (double)(rand() % g_maxNum) + 1.0;
				}
			}
		}
	}

	if (strcmp(g_init, "fast") == 0) {
		for (int i = 0; i < g_matrixSize; i++) {
			for (int j = 0; j < g_matrixSize; j++) {
				if (i == j) { //diagonal dominance
					matrix[i][j] = 5.0;
				}
				else {
					matrix[i][j] = 2.0;
				}
			}
		}
	}

	return matrix;
}

double **initMatrixPar() {
	double **matrix;// = new double*[g_matrixSize];
	hipMallocManaged(&matrix, g_matrixSize * sizeof(double *));

	for (int i = 0; i < g_matrixSize; i++) {
		hipMallocManaged(&matrix[i], g_matrixSize * sizeof(double));
	}

	if (strcmp(g_init, "rand") == 0) {
		for (int i = 0; i < g_matrixSize; i++) {
			for (int j = 0; j < g_matrixSize; j++) {
				if (i == j) { //diagonal dominance
					matrix[i][j] = (double)(rand() % g_maxNum) + 5.0;
				}
				else {
					matrix[i][j] = (double)(rand() % g_maxNum) + 1.0;
				}
			}
		}
	}

	if (strcmp(g_init, "fast") == 0) {
		for (int i = 0; i < g_matrixSize; i++) {
			for (int j = 0; j < g_matrixSize; j++) {
				if (i == j) { //diagonal dominance
					matrix[i][j] = 5.0;
				}
				else {
					matrix[i][j] = 2.0;
				}
			}
		}
	}

	return matrix;
}

double *initVectorBSeq() {
	double *vectorB = new double[g_matrixSize];

	for (int i = 0; i < g_matrixSize; i++) {
		vectorB[i] = 2.0;
	}

	return vectorB;
}

double *initVectorYSeq() {
	double* vectorY = new double[g_matrixSize];

	for (int i = 0; i < g_matrixSize; i++) {
		vectorY[i] = 1.0;
	}

	return vectorY;
}

double *initVectorBPar() {
	double *vectorB;
	hipMallocManaged(&vectorB, g_matrixSize * sizeof(double));

	for (int i = 0; i < g_matrixSize; i++) {
		vectorB[i] = 2.0;
	}

	return vectorB;
}

double *initVectorYPar() {
	double* vectorY;
	hipMallocManaged(&vectorY, g_matrixSize * sizeof(double));

	for (int i = 0; i < g_matrixSize; i++) {
		vectorY[i] = 1.0;
	}

	return vectorY;
}

void gaussSeq(double **matrix, int matrixSize, double *vectorB, double *vectorY) {
	/* Gaussian elimination algorithm, Algo 8.4 from Grama */
	for (int k = 0; k < matrixSize; k++) { /* Outer loop */
		for (int j = k + 1; j < matrixSize; j++) {
			matrix[k][j] = matrix[k][j] / matrix[k][k]; /* Division step */
		}
		vectorY[k] = vectorB[k] / matrix[k][k];
		matrix[k][k] = 1.0;

		for (int i = k + 1; i < matrixSize; i++) {
			for (int j = k + 1; j < matrixSize; j++) {
				matrix[i][j] = matrix[i][j] - matrix[i][k] * matrix[k][j]; /* Elimination step */
			}
			vectorB[i] = vectorB[i] - matrix[i][k] * vectorY[k];
			matrix[i][k] = 0.0;
		}
	}
}

__global__
void gaussPar1(double **matrix, int matrixSize, double *vectorB, double *vectorY, int k) {
	int threadIndex = threadIdx.x;
	int blockIndex = blockIdx.x;
	int blockSize = blockDim.x;
	int stride = gridDim.x * blockSize;

	for (int j = k + 1 + threadIndex + blockIndex * blockSize; j < matrixSize; j += stride) {
		matrix[k][j] = matrix[k][j] / matrix[k][k]; /* Division step */
		for (int i = k + 1; i < matrixSize; i++) {
			matrix[i][j] = matrix[i][j] - matrix[i][k] * matrix[k][j]; /* Elimination step */
		}
	}
}

void print(double **matrix, int matrixSize, double *vectorB, double *vectorY) {
	printf("Matrix A: \n");
	for (int i = 0; i < matrixSize; i++) {
		printf("[");
		for (int j = 0; j < matrixSize; j++) {
			printf(" %5.2f,", matrix[i][j]);
		}
		printf("]\n");
	}

	printf("Vector b: \n[");
	for (int i = 0; i < matrixSize; i++) {
		printf(" %5.2f,", vectorB[i]);
	}
	printf("]\n");

	printf("Vector y: \n[");
	for (int i = 0; i < matrixSize; i++) {
		printf(" %5.2f,", vectorY[i]);
	}
	printf("]\n\n");
}

bool compareMatrices(double **matrix1, double **matrix2, int matrixSize) {
	//Check if matrix1 has the same values as matrix2
	bool isSame = true;

	for (int i = 0; i < matrixSize; i++) {
		for (int j = 0; j < matrixSize; j++) {
			if (abs(matrix1[i][j] - matrix2[i][j]) > 0.000001) {
				isSame = false;
				i = matrixSize;
				j = matrixSize;
			}
		}
	}

	return isSame;
}

bool compareVectors(double *vector1, double *vector2, int vectorSize) {
	//Check if vector1 has the same values as vector2
	bool isSame = true;

	for (int i = 0; i < vectorSize; i++) {
		if (abs(vector1[i] - vector2[i]) > 0.000001) {
			isSame = false;
			i = vectorSize;
		}
	}

	return isSame;
}

int main() {
	time_point<Clock> m_start, m_end;

	g_matrixSize = 2048;
	g_maxNum = 15;
	//g_init = "rand";
	g_init = "fast";

	//Allocate
	double **seqMatrix = initMatrixSeq();
	double *seqVectorB = initVectorBSeq();
	double *seqVectorY = initVectorYSeq();

	if (g_matrixSize < 15) {
		print(seqMatrix, g_matrixSize, seqVectorB, seqVectorY);
	}

	m_start = Clock::now();
	gaussSeq(seqMatrix, g_matrixSize, seqVectorB, seqVectorY);
	m_end = Clock::now();
	int seqGaussTime = (int)duration_cast<milliseconds>(m_end - m_start).count();
	std::cout << "Sequential gauss elimination took " << seqGaussTime << " milliseconds.\n";

	if (g_matrixSize < 15) {
		print(seqMatrix, g_matrixSize, seqVectorB, seqVectorY);
	}

	//Allocate
	double **parMatrix = initMatrixPar();
	double *parVectorB = initVectorBPar();
	double *parVectorY = initVectorYPar();

	int numberOfBlocks = 32;
	int numberOfThreadsPerBlock = 512;

	m_start = Clock::now();
	for (int k = 0; k < g_matrixSize; k++) {
		gaussPar1 << <numberOfBlocks, numberOfThreadsPerBlock >> > (parMatrix, g_matrixSize, parVectorB, parVectorY, k);
	}
	hipDeviceSynchronize();

	for (int k = 0; k < g_matrixSize; k++) {
		parVectorY[k] = parVectorB[k] / parMatrix[k][k];
		parMatrix[k][k] = 1.0;

		for (int i = k + 1; i < g_matrixSize; i++) {
			parVectorB[i] = parVectorB[i] - parMatrix[i][k] * parVectorY[k];
			parMatrix[i][k] = 0.0;
		}
	}
	m_end = Clock::now();

	int parGaussTime = (int)duration_cast<milliseconds>(m_end - m_start).count();
	std::cout << "Parallell gauss elimination took " << parGaussTime << " milliseconds.\n";

	if (g_matrixSize < 15) {
		print(parMatrix, g_matrixSize, parVectorB, parVectorY);
	}

	std::cout << "Check if the matrices are the same: ";
	if (compareMatrices(seqMatrix, parMatrix, g_matrixSize)) {
		std::cout << "Yes\n";
	}
	else {
		std::cout << "No \n";
	}

	std::cout << "Check if the B-vectors are the same: ";
	if (compareVectors(seqVectorB, parVectorB, g_matrixSize)) {
		std::cout << "Yes\n";
	}
	else {
		std::cout << "No \n";
	}

	std::cout << "Check if the Y-vectors are the same: ";
	if (compareVectors(seqVectorY, parVectorY, g_matrixSize)) {
		std::cout << "Yes\n";
	}
	else {
		std::cout << "No \n";
	}

	//Deallocate seq memory
	for (int i = 0; i < g_matrixSize; i++) {
		delete[] seqMatrix[i];
	}
	delete[] seqMatrix;
	delete[] seqVectorB;
	delete[] seqVectorY;

	//Deallocate unified (par) memory
	for (int i = 0; i < g_matrixSize; i++) {
		hipFree(parMatrix[i]);
	}
	hipFree(parMatrix);
	hipFree(parVectorB);
	hipFree(parVectorY);

	getchar();
	return 0;
}